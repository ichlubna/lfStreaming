#include "hip/hip_runtime.h"
#include "perPixel.h"
#include "kernels.h"
#include <stdexcept>

PerPixel::PerPixel(glm::ivec2 res) : resolution{res.x, res.y}, pixelCount{static_cast<size_t>(res.x*res.y)}
{
    if(hipMallocPitch(reinterpret_cast<void**>(&result), &pitch, resolution.x, resolution.y+(resolution.y+1)/2) != hipSuccess)
    //pitch = 2048;
    //if(hipMalloc(reinterpret_cast<void**>(&result), pitch*(resolution.y+(resolution.y+1)/2)) != hipSuccess)
        throw std::runtime_error("Cannot allocate result frame for per pixel interpolation.");
};

PerPixel::Result PerPixel::interpolate(PerPixel::InputFrames input)
{
    std::vector<float2> inOffsets;
    for(const auto &o : input.offsets)
        inOffsets.push_back({o.x*input.aspect, o.y}); 
    PerPixelInterpolation::perPixel(input.frames, input.weights, inOffsets, input.pitches, reinterpret_cast<uint8_t*>(result), input.inverseWeightSum, resolution, pitch);
    return {pitch, reinterpret_cast<hipDeviceptr_t>(result)};
}
