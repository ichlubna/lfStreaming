
#include <hip/hip_runtime.h>
#include <stdint.h>

namespace Conversion
{
    constexpr bool FLOAT_YUV_CONVERSION{false};

    __device__ bool coordsOutside(int2 coords, int2 resolution)
    {
        return (coords.x >= resolution.x || coords.y >= resolution.y);
    }

    __device__ int2 getImgCoords()
    {
        int2 coords;
        coords.x = (threadIdx.x + blockIdx.x * blockDim.x);
        coords.y = (threadIdx.y + blockIdx.y * blockDim.y);
        return coords;
    }

    __device__ int linearCoords(int2 coords, int2 resolution)
    {
        int linearCoords = coords.y*resolution.x + coords.x;
        return linearCoords; 
    }

    class NV12Block
    {
        public:
        static constexpr int BLOCK_SIZE{4};
        //source: https://learn.microsoft.com/en-us/windows/win32/medfound/recommended-8-bit-yuv-formats-for-video-rendering
        static constexpr float4 yuvCoefsFloat{1.596027, -0.391762, -0.812968, 2.017232};
        static constexpr float yuvCommonCoefFloat{1.164383};
        static constexpr int4 yuvCoefsInt{409, -100, -208, 516};
        static constexpr int yuvCommonCoefInt{298};
        int linearCoordsY[4];
        int linearCoordsUV;
        int Y[BLOCK_SIZE];
        int U;
        int V;
        
        __device__ uchar4 RGBToRGBA8(int3 RGB)
        {
            return{ static_cast<unsigned char>(RGB.x),
                    static_cast<unsigned char>(RGB.y),
                    static_cast<unsigned char>(RGB.z),
                    255};
        }    
        
        __device__ void initCoords(int2 coords, int2 doubleCoords, int2 resolution, int2 halfResolution, int pitch)
        { 
            int linear = linearCoords(doubleCoords, {pitch,resolution.y});
            linearCoordsY[0] = linear;
            linearCoordsY[1] = linear+1;
            linear = linearCoords({doubleCoords.x, doubleCoords.y+1}, {pitch,resolution.y});
            linearCoordsY[2] = linear;
            linearCoordsY[3] = linear+1;
            linearCoordsUV = linearCoords({doubleCoords.x, coords.y}, {pitch,resolution.y});
        }

        __device__ void load(uint8_t *NV12, int pixelCount)
        { 
            for(int i=0; i<BLOCK_SIZE; i++)
                Y[i] = NV12[linearCoordsY[i]]; 
            auto UVplane = NV12+pixelCount;
            U = UVplane[linearCoordsUV];
            V = UVplane[linearCoordsUV+1];
        }
   
        __device__ int clamp8Bit(float value)
        {
            if(value > 255)
                return 255;
            if(value < 0)
                return 0;
            return value;
        }
 
        __device__ int3 YRuvGuvBuvToRGB(float Y, float Ruv, float Guv, float Buv)
        {
            int R = clamp8Bit(round(Y + Ruv));
            int G = clamp8Bit(round(Y + Guv));
            int B = clamp8Bit(round(Y + Buv));
            return {R, G, B};
        }
        
        __device__ int3 YRuvGuvBuvToRGB(int Y, int Ruv, int Guv, int Buv)
        {
            int R = clamp8Bit((Y + Ruv + 128) >> 8);
            int G = clamp8Bit((Y + Guv + 128) >> 8);
            int B = clamp8Bit((Y + Buv + 128) >> 8);
            return {R, G, B};
        }

        template <typename T>
        __device__ void storeRGBA(hipSurfaceObject_t RGBA, int2 blockCoords)
        {
            int UN = U-128;
            int VN = V-128;
            T ruv, buv, guv, coef;
            if constexpr (std::is_same<T, float>::value)
            {
                ruv = VN*yuvCoefsFloat.x; 
                buv = UN*yuvCoefsFloat.y + VN*yuvCoefsFloat.z; 
                guv = UN*yuvCoefsFloat.w;
                coef = yuvCommonCoefFloat;
            }
            else if constexpr (std::is_same<T, int>::value)
            {
                ruv = VN*yuvCoefsInt.x;
                buv = UN*yuvCoefsInt.y + VN*yuvCoefsInt.z;
                guv = UN*yuvCoefsInt.w;
                coef = yuvCommonCoefInt;
            }

            const int2 offsets[4]{ {0,0}, {0,1}, {1,0}, {1,1} };
            for(int i=0; i<BLOCK_SIZE; i++)
            {
                uchar4 data = RGBToRGBA8(YRuvGuvBuvToRGB((Y[i]-16)*coef, ruv, buv, guv));
                surf2Dwrite(data, RGBA, (blockCoords.x+offsets[i].x)*sizeof(uchar4), blockCoords.y+offsets[i].y, hipBoundaryModeClamp);
            }
        }
    };

    __global__ void NV12ToRGBAKernel(uint8_t *NV12, hipSurfaceObject_t RGBA, int2 resolution, int2 halfResolution, int pixelCount, int pitch)
    {
        int2 coords = getImgCoords();
        if(coordsOutside(coords, halfResolution))
            return;
        int2 doubleCoords{coords.x*2, coords.y*2};

        NV12Block block;
        block.initCoords(coords, doubleCoords, resolution, halfResolution, pitch);
        block.load(NV12, pixelCount);
        if constexpr (FLOAT_YUV_CONVERSION)
            block.storeRGBA<float>(RGBA, {doubleCoords.x, resolution.y-1-doubleCoords.y}); 
        else
            block.storeRGBA<int>(RGBA, {doubleCoords.x, resolution.y-1-doubleCoords.y}); 
    }
    
    void NV12ToRGBA(uint8_t *NV12, hipSurfaceObject_t RGBA, int2 resolution, int pitch)
    {
        constexpr dim3 WG_SIZE{16,16,1};
        dim3 wgCount{1+resolution.x/WG_SIZE.x/2, 1+resolution.y/WG_SIZE.y/2, 1};
        NV12ToRGBAKernel<<<wgCount, WG_SIZE, 0>>>(NV12, RGBA, resolution, {resolution.x/2, resolution.y/2}, pitch*resolution.y, pitch);
    }
}
