
#include <hip/hip_runtime.h>
#include <stdint.h>

namespace Conversion
{
    __device__ bool coordsOutside(int2 coords, int2 resolution)
    {
        return (coords.x >= resolution.x || coords.y >= resolution.y);
    }

    __device__ int2 getImgCoords()
    {
        int2 coords;
        coords.x = (threadIdx.x + blockIdx.x * blockDim.x);
        coords.y = (threadIdx.y + blockIdx.y * blockDim.y);
        return coords;
    }

    __device__ int linearCoords(int2 coords, int2 resolution)
    {
        int linearCoords = coords.y*resolution.x + coords.x;
        return linearCoords; 
    }

    class NV12Block
    {
        public:
        static constexpr int BLOCK_SIZE{4};
        static constexpr float2 RUV{0, 1.13983};
        static constexpr float2 GUV{-0.39465, -0.58060};
        static constexpr float2 BUV{2.03211, 0};
        int linearCoordsY[4];
        int linearCoordsUV;
        float Y[BLOCK_SIZE];
        float U;
        float V;
        
        __device__ uchar4 RGBToRGBA8(float3 RGB)
        {
            return{ static_cast<unsigned char>(RGB.x),
                    static_cast<unsigned char>(RGB.y),
                    static_cast<unsigned char>(RGB.z),
                    255};
        }    
        
        __device__ void initCoords(int2 coords, int2 doubleCoords, int2 resolution, int2 halfResolution, int pitch)
        { 
            int linear = linearCoords(doubleCoords, {pitch,resolution.y});
            linearCoordsY[0] = linear;
            linearCoordsY[1] = linear+1;
            linear = linearCoords({doubleCoords.x, doubleCoords.y+1}, {pitch,resolution.y});
            linearCoordsY[2] = linear;
            linearCoordsY[3] = linear+1;
            linearCoordsUV = linearCoords({doubleCoords.x, coords.y+1}, {pitch,resolution.y});
            //linearCoordsUV = linearCoords({coords.x, coords.y}, {pitch, halfResolution.y});
        }

        __device__ void load(uint8_t *NV12, int pixelCount)
        { 
            for(int i=0; i<BLOCK_SIZE; i++)
                Y[i] = NV12[linearCoordsY[i]]; 
            auto UVplane = NV12+pixelCount;
            U = UVplane[linearCoordsUV];
            V = UVplane[linearCoordsUV+1];

        }

        __device__ void storeRGBA(hipSurfaceObject_t RGBA, int2 blockCoords)
        {
            float ruv = U*RUV.x + V*RUV.y; 
            float guv = U*GUV.x + V*GUV.y; 
            float buv = U*BUV.x + V*BUV.y; 

            const int2 offsets[4]{ {0,0}, {0,1}, {1,0}, {1,1} };
            for(int i=0; i<BLOCK_SIZE; i++)
            {
                int2 coords{blockCoords.x+offsets[i].x, blockCoords.y+offsets[i].y};
                uchar4 data = RGBToRGBA8({Y[i]+ruv, Y[i]+guv, Y[i]+buv});
                //data = RGBToRGBA8({Y[i], Y[i], Y[i]});
                surf2Dwrite(data, RGBA, (blockCoords.x+offsets[i].x)*sizeof(uchar4), blockCoords.y+offsets[i].y, hipBoundaryModeClamp);
            }
        }
    };

    __global__ void NV12ToRGBAKernel(uint8_t *NV12, hipSurfaceObject_t RGBA, int2 resolution, int2 halfResolution, int pixelCount, int pitch)
    {
        int2 coords = getImgCoords();
        if(coordsOutside(coords, halfResolution))
            return;
        int2 doubleCoords{coords.x*2, coords.y*2};

        NV12Block block;
        block.initCoords(coords, doubleCoords, resolution, halfResolution, pitch);
        block.load(NV12, pixelCount);
        block.storeRGBA(RGBA, {doubleCoords.x, resolution.y-1-doubleCoords.y}); 
    }
    
    void NV12ToRGBA(uint8_t *NV12, hipSurfaceObject_t RGBA, int2 resolution, int pitch)
    {
        constexpr dim3 WG_SIZE{16,16,1};
        dim3 wgCount{1+resolution.x/WG_SIZE.x/2, 1+resolution.y/WG_SIZE.y/2, 1};
        NV12ToRGBAKernel<<<wgCount, WG_SIZE, 0>>>(NV12, RGBA, resolution, {resolution.x/2, resolution.y/2}, pitch*resolution.y, pitch);
    }
}
