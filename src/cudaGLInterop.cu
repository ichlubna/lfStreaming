#include <stdexcept>
#include "cudaGLInterop.h"
#include "kernels.cu"

CudaGLInterop::~CudaGLInterop()
{
    unset();
}

void CudaGLInterop::unset()
{
    if(registered)
        if(hipGraphicsUnregisterResource(graphicsResource) != hipSuccess)
            throw std::runtime_error("Cannot unregister graphics resource.");
    registered = false;
}

void CudaGLInterop::setTexture(GLuint inputTexture, glm::ivec2 textureResolution)
{
    unset();
    textureGL = inputTexture;
    resolution = textureResolution;
    if(hipGraphicsGLRegisterImage(&graphicsResource, textureGL, GL_TEXTURE_2D, hipGraphicsRegisterFlagsWriteDiscard) != hipSuccess)
        throw std::runtime_error("Cannot register graphics resource.");
}

void CudaGLInterop::copyData(hipDeviceptr_t input)
{
    hipGraphicsMapResources(1, &graphicsResource);
    hipArray_t output;
    hipGraphicsSubResourceGetMappedArray(&output, graphicsResource, 0, 0);
    hipResourceDesc resourceDesc;
    resourceDesc.resType = hipResourceTypeArray;
    resourceDesc.res.array.array = output;
    hipSurfaceObject_t surface;
    hipCreateSurfaceObject(&surface, &resourceDesc); 
    Conversion::NV12ToRGBA(reinterpret_cast<uint8_t*>(input), surface, {resolution.x, resolution.y});
    hipDestroySurfaceObject(surface);
    hipGraphicsUnmapResources(1, &graphicsResource);
    hipStreamSynchronize(0); 
}
